#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "HnHelper.cuh"
#include "HnGPU.cuh"

#include <chrono>
#include <iostream>

void argTest(std::string query, std::string data) {
    int* test;
    hipDeviceReset();

    //Warmup the device!
    for (int i = 0; i < 5; i++) {
        hipMalloc(&test, UPPERMEMLIMIT);
        HnSetup::dummy <<<1, 1 >>> ();
        hipFree(test);
    }

#ifdef NAMEDATA
    std::cout << "\n\n----------------------------------------------------------------------\n\n";
    std::cout << "       " << "GFSM: " << query << " --> " << data;
    std::cout << "\n\n----------------------------------------------------------------------\n\n";
#endif

    csv_printf("\n%s,%s,", query.substr(query.find_last_of("/") + 1).c_str(), data.substr(data.find_last_of("/")+1).c_str());

    HnSetup::preinit(MEMLIMIT, MAXSOLNSIZE, MAXSCANSIZE);
    auto start = std::chrono::steady_clock::now();

    CCSR::CCSRStagger queryStagger;
    CCSR::CCSRGraph queryGraph = fileParse(query, false, &queryStagger);
    //print(queryGraph);

    CCSR::CCSRStagger dataStagger;
    CCSR::CCSRGraph dataGraph = fileParse(data, false, &dataStagger);
    //print(dataGraph);

    HnSetup::solve(queryGraph, dataGraph, queryStagger, dataStagger);

    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    info_printf("\nEnd to End Time: %fs\n", elapsed_seconds.count());
    csv_printf("%fs,", elapsed_seconds.count());
}

void nullArgTest() {
    argTest("query/_5_road5.g", "data/roadNet-PA.g"); //"data/roadNet-PA.g"
    //argTest("query/_3_triangle.g", "data/cit-Patents-Sorted.g"); //"data/roadNet-PA.g"
    //argTest("query/DBpedia-Queries/_2_query_1.g", "data/DBPedia.g"); //"data/roadNet-PA.g"
    //argTest("query/roadNet-PA-Queries/_11_query_15.g", "data/roadNet-PA-Labels.g");
}

int main(int argc, char* argv[])
{
    if (argc == 3) {
        argTest(std::string(argv[1]), std::string(argv[2]));
    }
    else {
        nullArgTest();
    }

}
